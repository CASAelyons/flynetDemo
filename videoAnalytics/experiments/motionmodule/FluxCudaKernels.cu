#include "hip/hip_runtime.h"

/*///////////////////////////////////////////////////////////////////////*/
//  	Hadi AliAkbarpour
//
//  For more information, contact:
//      Dr. Hadi AliAkbarpour, hd.akbarpour@gmail.com, aliakbarpourh@missouri.edu ,
//		Dr. Filiz Bunyak,
//      Prof. K. Palaniappan palaniappank@missouri.edu,
//      329 Engineering Building West
//      University of Missouri-Columbia
//      Columbia, MO 65211
//
//
/*///////////////////////////////////////////////////////////////////////////*/

#include "FluxCudaKernels.h"

const int BLOCK_SIZE1 = 16;
const int BLOCK_SIZE2 = 16;

const int border = 30 ;

__constant__ ImageProp ip ;

__constant__ float td1[20] ;
__constant__ float td2[20] ;
__constant__ int td1_size ;
__constant__ float tavg[20] ;
__constant__ int tavg_size ;

__global__
void hello(char *a, int *b)
{
    a[threadIdx.x] += b[threadIdx.x];
}

void testHello(char* a,int *b, int csize, int isize)
{
    char *ad;
    int *bd;
    hipMalloc( (void**)&ad, csize );
    hipMalloc( (void**)&bd, isize );
    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

    dim3 dimBlock( BLOCK_SIZE1, 1 );
    dim3 dimGrid( 1, 1 );
    hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
    hipFree( ad );
    hipFree( bd );
}

__global__ void addImages_kernel(float * a, float * b, float * c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
#if 1
    int w= ip.w ;
    int h= ip.h ;
    int stride = ip.stride ;
#endif
#if 0
    int w= 1000;
    int h= 1000 ;
    int pitch = 1000 ;
#endif
    if ((i >= 0) && (j >= 0) && (i < (w-1)) && (j < (h-1)))
    {
        float *p_a = a + j*stride + i ;
        float *p_b = b + j*stride + i ;
        float *p_c = c + j*stride + i ;
        *p_c = *p_a + *p_b ;
    }
}
void addImages(float * a, float * b, float * c,int w, int h)
{
#if 0
    dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    dim3 numBlocks(1000/threadsPerBlock.x,  /* for instance 512/8 = 64*/
                   1000/threadsPerBlock.y);
#endif
#if 1
    dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    dim3 numBlocks(w/threadsPerBlock.x,  /* for instance 512/8 = 64*/
                   h/threadsPerBlock.y);
#endif
    addImages_kernel<<<numBlocks, threadsPerBlock>>>(a, b , c);
}

void FluxCudaKernelsInit(ImageProp *imgProp , float *td1_ , float *td2_, int td1_size_ , float *tavg_, int tavg_size_)
{
    hipMemcpyToSymbol(HIP_SYMBOL(ip), imgProp, sizeof(ImageProp));
    hipMemcpyToSymbol(HIP_SYMBOL(td1), td1_, sizeof(float)*td1_size_);
    hipMemcpyToSymbol(HIP_SYMBOL(td2), td2_, sizeof(float)*td1_size_);
    hipMemcpyToSymbol(HIP_SYMBOL(td1_size), &td1_size_, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(tavg), tavg_, sizeof(float)*tavg_size_);
    hipMemcpyToSymbol(HIP_SYMBOL(tavg_size), &tavg_size_, sizeof(int));
}

////////////////////////////////////////////////////////
__global__ void TEST(int width,int height, int pitch, float *Ix[] , float *Iy[], float *Is[] , float *Mxt, float *Myt, float *Mt, float *Mtt, float *Mtrace, float *MtraceST, int masterFrameIndex)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    pitch /= sizeof(float) ; // making it proper for later pointer usage
    int idx = j*pitch + i;

    float v_Mxt = 0.0f;
    float v_Myt = 0.0f;
    float v_Mt = 0.0f;
    float v_Mtt = 0.0f;

    if ((i >= 0) && (j >= 0) && (i < 100) && (j < 100))
        //if ((i >= border) && (j >= border) && (i < (width-border-1)) && (j < (height-border-1)))
    {
        //float *p_a = a + j*stride + i ;
        float *p_Mxt = Mxt + idx;
        float *p_Myt = Myt + idx;
        float *p_Mt  = Mt + idx;
        float *p_Mtt = Mtt + idx;
        float *p_Mtrace = Mtrace + idx;
        float *p_MtraceST = MtraceST + idx;
        float *p_Ix ;
        float *p_Iy ;
        float *p_Is ;

        *p_Mxt = 0 ;
        *p_Myt = 0 ;
        *p_Mt = 0 ;
        *p_Mtt = 0 ;

        //for(int k=0 ; k<td1_size ; k++ ) {
        for(int k=0 ; k<5 ; k++ ) {
            p_Ix = Ix[k] + idx;
            p_Iy = Iy[k] + idx;
            p_Is = Is[k] + idx;
            (*p_Mxt) +=  (*p_Ix)*td1[k] ;
            (*p_Myt) +=  (*p_Iy)*td1[k] ;
            (*p_Mt) +=  (*p_Is)*td1[k] ;
            (*p_Mtt) +=  (*p_Is)*td2[k] ;

            v_Mxt +=  (*p_Ix)*td1[k] ;
            v_Myt +=  (*p_Iy)*td1[k] ;
            v_Mt +=  (*p_Is)*td1[k] ;
            v_Mtt +=  (*p_Is)*td2[k] ;

            (*p_Ix) = (float)td1_size ;
            //(*p_Ix) = 99.0 ;
        }
        //*p_Mtrace = v_Mxt ;
        // for flux tensor
        //*p_Mtrace = sqrt((*p_Mxt)*(*p_Mxt) + (*p_Myt)*(*p_Myt) + (*p_Mtt)*(*p_Mtt) );
        *p_Mtrace = sqrt((v_Mxt)*(v_Mxt) + (v_Myt)*(v_Myt) + (v_Mtt)*(v_Mtt) );

        // for 3D structure tensor
        p_Ix = Ix[masterFrameIndex] + idx;
        p_Iy = Iy[masterFrameIndex] + idx;
        //*p_MtraceST = sqrt((*p_Ix)*(*p_Ix) + (*p_Iy)*(*p_Iy) + (*p_Mt)*(*p_Mt) );
        *p_MtraceST = sqrt((*p_Ix)*(*p_Ix) + (*p_Iy)*(*p_Iy) + (v_Mt)*(v_Mt) );
    }
}

////////////////////////////////////////////////////////
__global__ void FluxDerivKernel(int width,int height, int pitch, float *Ix[] , float *Iy[], float *Is[] , float *Mxt, float *Myt, float *Mt, float *Mtt, float *Mtrace, float *MtraceST, int masterFrameIndex)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    pitch /= sizeof(float) ; // making it proper for later pointer usage
    int idx = j*pitch + i;

    float v_Mxt = 0.0f;
    float v_Myt = 0.0f;
    float v_Mt = 0.0f;
    float v_Mtt = 0.0f;

    //if ((i >= 50) && (j >= 50) && (i < 100) && (j < 100))
    if ((i >= border) && (j >= border) && (i < (width-border-1)) && (j < (height-border-1)))
    {
        //float *p_a = a + j*stride + i ;
        float v_Ix, v_Iy, v_Is ;

        for(int k=0 ; k<td1_size ; k++ ) {
            v_Ix = (Ix[k])[idx];
            v_Iy = (Iy[k])[idx];
            v_Is = (Is[k])[idx];

            v_Mxt +=  v_Ix*td1[k] ;
            v_Myt +=  v_Iy*td1[k] ;
            v_Mt +=  v_Is*td1[k] ;
            v_Mtt +=  v_Is*td2[k] ;
        }

        Mxt[idx] = v_Mxt;
        Myt[idx] = v_Myt;
        Mt[idx] = v_Mt;
        Mtt[idx] = v_Mtt;

        // for flux tensor
        Mtrace[idx] = sqrt((v_Mxt)*(v_Mxt) + (v_Myt)*(v_Myt) + (v_Mtt)*(v_Mtt) );

        // for 3D structure tensor
        v_Ix = (Ix[masterFrameIndex])[idx];
        v_Iy = (Iy[masterFrameIndex])[idx];
        MtraceST[idx] = sqrt(v_Ix*v_Ix + v_Iy*v_Iy + v_Mt*v_Mt );
    }
}

void FluxDerivByKernel(int width, int height, int pitch, int masterFrameIndex, float *Ix[] , float *Iy[], float *Is[] , float *Mxt, float *Myt, float *Mt, float *Mtt, float *Mtrace, float *MtraceST)
{
    static bool first=true ;
    static dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    static dim3 numBlocks(width/threadsPerBlock.x,  height/threadsPerBlock.y);
    static int nt_deriv ;
    if(first)
        hipMemcpyFromSymbol(&nt_deriv,HIP_SYMBOL(td1_size), sizeof(int));

    //printf("td1_size is %d\n",nt_deriv) ;

    static float **d_IxData ;
    static float **d_IyData ;
    static float **d_IsData ;
    if(first) {
        cudaMemErrChk2(hipMalloc((void**)&d_IxData, nt_deriv * sizeof(float *)));
        cudaMemErrChk2(hipMalloc((void**)&d_IyData, nt_deriv * sizeof(float *)));
        cudaMemErrChk2(hipMalloc((void**)&d_IsData, nt_deriv * sizeof(float *)));
    }

    //	for(int i=0 ; i<nt_deriv ; i++) {
    //		hipMemcpy(&(d_IxData[i]), &(Ix[i]), sizeof(float *), hipMemcpyHostToDevice);//copy child pointer to device
    //		hipMemcpy(&(d_IyData[i]), &(Iy[i]), sizeof(float *), hipMemcpyHostToDevice);//copy child pointer to device
    //		hipMemcpy(&(d_IsData[i]), &(Is[i]), sizeof(float *), hipMemcpyHostToDevice);//copy child pointer to device
    //	}

    hipMemcpy(d_IxData, Ix, nt_deriv*sizeof(float *), hipMemcpyHostToDevice);//copy child pointer to device
    hipMemcpy(d_IyData, Iy, nt_deriv*sizeof(float *), hipMemcpyHostToDevice);//copy child pointer to device
    hipMemcpy(d_IsData, Is, nt_deriv*sizeof(float *), hipMemcpyHostToDevice);//copy child pointer to device

    FluxDerivKernel<<<numBlocks, threadsPerBlock>>>(width, height, pitch, d_IxData , d_IyData , d_IsData , Mxt , Myt, Mt , Mtt, Mtrace, MtraceST, masterFrameIndex);
    //TEST<<<numBlocks, threadsPerBlock>>>(width, height, pitch, d_IxData , d_IyData , d_IsData , Mxt , Myt, Mt , Mtt, Mtrace, MtraceST, masterFrameIndex);

    first = false ;
    return ;
    //ssd These lines are unreachable	hipFree(d_IxData) ;
    //ssd These lines are unreachable	hipFree(d_IyData) ;
    //ssd These lines are unreachable	hipFree(d_IsData) ;
}
//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
__global__ void FluxTempAVGKernel(int width,int height, int pitch, float *TraceSumXY[] , float *TraceSTSumXY[], float *FluxTrace, float * STTrace, float d_denominator)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    pitch /= sizeof(float) ; // making it proper for later pointer usage
    int idx = j*pitch + i;
    //if ((i >= 0) && (j >= 0) && (i < (width-1)) && (j < (height-1)))
    if ((i >= border) && (j >= border) && (i < (width-border-1)) && (j < (height-border-1)))
    {
/*        float *p_FluxTrace = FluxTrace + idx ;
        float *p_STTrace = STTrace + idx ;
        float *p_TraceSumXY ;
        float *p_TraceSTSumXY ;      
        *p_FluxTrace = 0 ;
        *p_STTrace = 0 ;
*/
        float v_FluxTrace = 0.0f;
        float v_STTrace = 0.0;

        for(int k=0 ; k<tavg_size ; k++ ) {
 /*           p_TraceSumXY = TraceSumXY[k] + idx ;
            (*p_FluxTrace) += (*p_TraceSumXY)*tavg[k] ;

            p_TraceSTSumXY = TraceSTSumXY[k] + idx ;
            (*p_STTrace) += (*p_TraceSTSumXY)*tavg[k] ;
 */
             v_FluxTrace += (TraceSumXY[k])[idx] * tavg[k] ;

             v_STTrace += (TraceSTSumXY[k])[idx] * tavg[k] ;
        }
/*        (*p_FluxTrace) /= d_denominator ;
        (*p_STTrace) /= d_denominator ;
        */
        FluxTrace[idx] = v_FluxTrace / d_denominator;
        STTrace[idx] = v_STTrace / d_denominator;
    }
}
void FluxTempAVGByKernel(int width,int height, int pitch, float *TraceSumXY[] , float *TraceSTSumXY[], float *FluxTrace, float * STTrace, float denominator)
{
    static bool first=true ;
    static dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    static dim3 numBlocks(width/threadsPerBlock.x,  height/threadsPerBlock.y);
    static int nt_avg ;

    static float **d_TraceSumXY ;
    static float **d_TraceSTSumXY;
    //static float *d_denominator ;
    if(first) {
        hipMemcpyFromSymbol(&nt_avg,HIP_SYMBOL(tavg_size), sizeof(int));
        cudaMemErrChk2(hipMalloc((void**)&d_TraceSumXY, nt_avg * sizeof(float *)));
        cudaMemErrChk2(hipMalloc((void**)&d_TraceSTSumXY, nt_avg * sizeof(float *)));

        //cudaMemErrChk2(hipMalloc((void**)&d_denominator,  sizeof(float)));
        //hipMemcpy(d_denominator , &denominator, sizeof(float), hipMemcpyHostToDevice);
    }
    hipMemcpy(d_TraceSumXY , TraceSumXY, nt_avg*sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(d_TraceSTSumXY , TraceSTSumXY, nt_avg*sizeof(float *), hipMemcpyHostToDevice);

    FluxTempAVGKernel<<<numBlocks, threadsPerBlock>>>(width, height, pitch,d_TraceSumXY,d_TraceSTSumXY,FluxTrace,STTrace, denominator) ;

    first = false ;
    return ;
}

//--------------------------------------------
// -----  detection--computeDifference -------
//--------------------------------------------
__global__ void computeDifferenceKernel(int width,int height, int pitch, float *sum_head , float *Ei, float *E, float *BG, float *E2d, float *D)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int offset ;
    pitch /= sizeof(float) ; // making it proper for later pointer usage
    //static float ww=0 ; //sa 1,25
    //if ((i >= 0) && (j >= 0) && (i <= (width-1)) && (j <= (height-1)))
    if ((i >= border) && (j >= border) && (i < (width-border-1)) && (j < (height-border-1)))
    {
        offset =  j*pitch + i ;
        float *p_sum_head = sum_head + offset ;
        float *p_Ei = Ei  + offset  ;
        //ssd        float *p_E = E + offset  ;
        float *p_BG = BG + offset  ;
        float *p_D = D + offset  ;
        float *p_E2d = E2d + offset  ;
        //*p_D = (*p_Ei) - (*p_E2d) - ww * (*p_BG) ; //sa
        *p_D = (*p_Ei) - (*p_E2d)  ;
        if((*p_D)<0)
            *p_D = 0 ;
        (*p_sum_head) += *p_D ;
    }
}
void computeDifference(int width, int height, int pitch, float *sum_head, int *count, float *Ei, float *E, float *BG, float *E2d, float *D, int type)
{
    if(type!=0) //not supported
        return ;
    static dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    static dim3 numBlocks(width/threadsPerBlock.x,  height/threadsPerBlock.y);
    computeDifferenceKernel<<<numBlocks, threadsPerBlock>>>(width, height, pitch,sum_head,Ei,E,BG,E2d,D) ;
    (*count)++ ;
}
//----------------------------------------------------------------------------------------------
__global__ void computeDifferenceKernel_color( float *sum_head_r, float *sum_head_g, float *sum_head_b,
                                               float *Ei_r,float *Ei_g,float *Ei_b, float *E_r,float *E_g,float *E_b, float *BG, float *D)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int offset ;
    int w= ip.w ;
    int h= ip.h ;
    int stride = ip.stride ;
    static float ww=1.25 ;
    const float EPS=0.0001 ;
    if ((i >= 0) && (j >= 0) && (i <= (w-1)) && (j <= (h-1)))
    {
        offset =  j*stride + i ;
        float *p_sum_head_r = sum_head_r + offset ;
        float *p_sum_head_g = sum_head_g + offset ;
        //ssd        float *p_sum_head_b = sum_head_b + offset ;

        float *p_Ei_r = Ei_r  + offset  ;
        float *p_Ei_g = Ei_g  + offset  ;
        float *p_Ei_b = Ei_b  + offset  ;

        float *p_E_r = E_r + offset  ;
        float *p_E_g = E_g + offset  ;
        float *p_E_b = E_b + offset  ;
        //ssd        float *p_D = D + offset  ;

        float *p_BG = BG + offset  ;

        float si = (*p_Ei_r) +  (*p_Ei_g) + (*p_Ei_b) + EPS ;
        float s = (*p_E_r) +  (*p_E_g) + (*p_E_b) + EPS  ;

        //DDD
        //*p_D = *p_Ei_g ;


#if 1

        float D_r = 150*abs((*p_Ei_r)/si-(*p_E_r)/s) - ww*(*p_BG) ;
        float D_g = 150*abs((*p_Ei_g)/si-(*p_E_g)/s) - ww*(*p_BG) ;


        //        double D_r = 150*abs((*p_Ei_r)/si-(*p_E_r)/s) - ww*(*p_BG) ;
        //        double D_g = 150*abs((*p_Ei_g)/si-(*p_E_g)/s) - ww*(*p_BG) ;

#else
        (*p_Ei_r) /= si ;
        (*p_Ei_g) /= si ;
        (*p_Ei_b) = 0 ;

        (*p_E_r) /= s ;
        (*p_E_g) /= s ;
        (*p_E_b) = 0 ;

        double D_r = 150*abs((*p_Ei_r)-(*p_E_r)) - ww*(*p_BG) ;
        double D_g = 150*abs((*p_Ei_g)-(*p_E_g)) - ww*(*p_BG) ;
#endif

        if( D_r < 0 )
            D_r = 0 ;
        if( D_g < 0 )
            D_g = 0 ;



        (*p_sum_head_r) += D_r ;
        (*p_sum_head_g) += D_g ;


    }
}
void computeDifference_Color(int w, int h,
                             float *sum_head_r, float *sum_head_g, float *sum_head_b,
                             int *count,
                             float *Ei_r, float *Ei_g, float *Ei_b,
                             float *E_r, float *E_g, float *E_b,
                             float *BG,float *D)
{
    static dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    static dim3 numBlocks(w/threadsPerBlock.x,  h/threadsPerBlock.y);
    computeDifferenceKernel_color<<<numBlocks, threadsPerBlock>>>(sum_head_r,sum_head_g,sum_head_b,Ei_r,Ei_g,Ei_b,E_r,E_g,E_b,BG,D) ;
    (*count)++ ;
}

//--------------------------------------------
// -----  detection--checkPersistency -------
//--------------------------------------------
__global__ void checkPersistencyKernel(int width, int height, int pitch, float *E,float *sum_head,float *sum_tail, float *mask, float *pm, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int offset ;
    pitch /= sizeof(float) ; // making it proper for later pointer usage
    //if ((i >= 0) && (j >= 0) && (i <= (width-1)) && (j <= (height-1)))
    if ((i >= border) && (j >= border) && (i < (width-border-1)) && (j < (height-border-1)))
    {
        offset =  j*pitch + i ;
        //ssd        float *p_E = E + offset  ;
        float *p_sum_head = sum_head + offset ;
        float *p_sum_tail = sum_tail + offset ;
        float *p_pm = pm + offset  ;
        float *p_mask = mask + offset  ;
        *p_pm = ((*p_sum_head) - (*p_sum_tail)) / n ;
        *p_mask = *p_pm > 10 ? 255 : 0 ;
    }
}
void checkPersistency(int width, int height, int pitch, float *E, float *sum_head,float *sum_tail, float *mask, float *pm,int count_head, int count_tail,int type)
{
    if(type!=0) //not supported
        return ;
    int n=count_head-count_tail+1 ;
    static dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    static dim3 numBlocks(width/threadsPerBlock.x,  height/threadsPerBlock.y);
    checkPersistencyKernel<<<numBlocks, threadsPerBlock>>>(width, height, pitch, E, sum_head, sum_tail, mask, pm, n);
}
//------------------------------------------------------------------------------
__global__ void checkPersistencyKernel_Color(float *E_r,float *E_g,float *E_b,float *sum_head_r,float *sum_head_g,float *sum_head_b,float *sum_tail_r,float *sum_tail_g,float *sum_tail_b, float *mask, float *pm, int n , float *D)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int offset ;
    int w= ip.w ;
    int h= ip.h ;
    int stride = ip.stride ;
    if ((i >= 0) && (j >= 0) && (i <= (w-1)) && (j <= (h-1)))
    {
        offset =  j*stride + i ;

        //ssd        float *p_D = D + offset  ;

        float *p_E_r = E_r + offset  ;
        float *p_E_g = E_g + offset  ;
        float *p_E_b = E_b + offset  ;

        float *p_sum_head_r = sum_head_r + offset ;
        float *p_sum_head_g = sum_head_g + offset ;
        float *p_sum_head_b = sum_head_b + offset ;

        float *p_sum_tail_r = sum_tail_r + offset ;
        float *p_sum_tail_g = sum_tail_g + offset ;
        float *p_sum_tail_b = sum_tail_b + offset ;

        float *p_pm = pm + offset  ;
        float *p_mask = mask + offset  ;

        // picking the max in r,g,b
        float tmp_r = ((*p_sum_head_r) - (*p_sum_tail_r)) / n ;
        float tmp_g = ((*p_sum_head_g) - (*p_sum_tail_g)) / n ;
        float tmp_b = ((*p_sum_head_b) - (*p_sum_tail_b)) / n ;
        float tmp_final = tmp_r >= tmp_g ? tmp_r : tmp_g ;
        if(tmp_final<tmp_b)
            tmp_final = tmp_b ;

        //DDD
        //*p_D = tmp_r ;

        (*p_pm) = tmp_final ;

        float s = (*p_E_r) + (*p_E_g) + (*p_E_b) ;
        float B = s/(3*255) ;
        (*p_pm) *= B ;

        //debug
        //*p_pm = tmp_final ;
        //*p_pm = ((*p_sum_head_r) - (*p_sum_tail_r)) / n ;

        *p_mask = *p_pm > 10 ? 255 : 0 ;
    }
}
void checkPersistency_Color(int w, int h,float *E_r,float *E_g,float *E_b,float *sum_head_r,float *sum_head_g,float *sum_head_b,float *sum_tail_r,float *sum_tail_g,float *sum_tail_b, float *mask, float *pm,int count_head, int count_tail,float *D)
{
    int n=count_head-count_tail+1 ;
    static dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    static dim3 numBlocks(w/threadsPerBlock.x,  h/threadsPerBlock.y);
    checkPersistencyKernel_Color<<<numBlocks, threadsPerBlock>>>(E_r,E_g,E_b,sum_head_r,sum_head_g,sum_head_b,sum_tail_r,sum_tail_g,sum_tail_b,mask,pm,n,D);
}
//--------------------------------------------
// -----  detection--accumFlux -------
//--------------------------------------------
__global__ void accumFluxKernel_Grey(int width,int height, int pitch, float *flux,float *E,float *I, int n_frames , float *out_flux_mean, float *out_E_mean, float *out_I_mean)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int offset ;
    pitch /= sizeof(float) ; // making it proper for later pointer usage
    //if ((i >= 0) && (j >= 0) && (i <= (width-1)) && (j <= (height-1)))
    if ((i >= border) && (j >= border) && (i < (width-border-1)) && (j < (height-border-1)))
    {
        offset =  j*pitch + i ;
        float *p_flux = flux + offset  ;
        float *p_E = E + offset  ;
        //float *p_I = I + offset  ;
        float *p_out_flux_mean = out_flux_mean + offset  ;
        float *p_out_E_mean = out_E_mean + offset  ;
        //ssd        float *p_out_I_mean = out_I_mean + offset  ;
        (*p_out_flux_mean) += (*p_flux)/n_frames ;
        (*p_out_E_mean) += (*p_E)/n_frames ;
        //*p_out_I_mean += (*p_I)/n_frames ;
    }
}
void accumFlux(int width,int height, int pitch, float *flux,float *E,float *I, int n_frames , float *out_flux_mean, float *out_E_mean, float *out_I_mean)
{
    static dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    static dim3 numBlocks(width/threadsPerBlock.x,  height/threadsPerBlock.y);
    accumFluxKernel_Grey<<<numBlocks, threadsPerBlock>>>(width, height, pitch,flux,E,I,n_frames,out_flux_mean,out_E_mean,out_I_mean);
}
//--------------------------------------------------------------------------------------------------------------------------------------------------------------
__global__ void accumFluxKernel_Color(float *flux,float *E,float *Ir,float *Ig,float *Ib, int n_frames , float *out_flux_mean, float *out_E_mean, float *out_Ir_mean,float *out_Ig_mean,float *out_Ib_mean)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int offset ;
    int w= ip.w ;
    int h= ip.h ;
    int stride = ip.stride ;
    if ((i >= 0) && (j >= 0) && (i <= (w-1)) && (j <= (h-1)))
    {
        offset =  j*stride + i ;
        float *p_flux = flux + offset  ;
        float *p_E = E + offset  ;
        float *p_Ir = Ir + offset  ;
        float *p_Ig = Ig + offset  ;
        float *p_Ib = Ib + offset  ;
        float *p_out_flux_mean = out_flux_mean + offset  ;
        float *p_out_E_mean = out_E_mean + offset  ;
        float *p_out_Ir_mean = out_Ir_mean + offset  ;
        float *p_out_Ig_mean = out_Ig_mean + offset  ;
        float *p_out_Ib_mean = out_Ib_mean + offset  ;
        (*p_out_flux_mean) += (*p_flux)/n_frames ;
        (*p_out_E_mean) += (*p_E)/n_frames ;
        (*p_out_Ir_mean) += (*p_Ir)/n_frames ;
        (*p_out_Ig_mean) += (*p_Ig)/n_frames ;
        (*p_out_Ib_mean) += (*p_Ib)/n_frames ;
    }
}

void accumFlux_Color(int w, int h,float *flux,float *E,
                     float *Ir,float *Ig,float *Ib,
                     int n_frames , float *out_flux_mean, float *out_E_mean,
                     float *out_Ir_mean,float *out_Ig_mean,float *out_Ib_mean)
{
    static dim3 threadsPerBlock( BLOCK_SIZE1, BLOCK_SIZE2 );
    static dim3 numBlocks(w/threadsPerBlock.x,  h/threadsPerBlock.y);
    accumFluxKernel_Color<<<numBlocks, threadsPerBlock>>>(flux,E,Ir,Ig,Ib,n_frames,out_flux_mean,out_E_mean,out_Ir_mean,out_Ig_mean,out_Ib_mean);
}
